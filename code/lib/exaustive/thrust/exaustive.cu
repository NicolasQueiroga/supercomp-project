#include "hip/hip_runtime.h"
#include "exaustive.hpp"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/iterator/counting_iterator.h>

__device__ bool Thrust::movieIsValid(const Movie *movie)
{
    this->movieScheduleBitset->reset();
    for (short j = movie->startTime; j < movie->endTime; j++)
        this->movieScheduleBitset->set(j);

    if ((!(*this->movieScheduleBitset & *this->agendaBitset).any() &&
         this->maxMoviesPerCat[movie->category - 1] > 0) &&
        (movie->startTime < movie->endTime && movie->startTime >= 0 && movie->endTime < 24))
        return 1;
    return 0;
}

struct movie_validity_checker : public thrust::unary_function<int, bool>
{
    const Movie *moviesList;
    int *agendaBitset;
    int *movieScheduleBitset;
    int *maxMoviesPerCat;

    __device__ movie_validity_checker(const Movie *_moviesList, int *_agendaBitset, int *_movieScheduleBitset, int *_maxMoviesPerCat)
        : moviesList(_moviesList), agendaBitset(_agendaBitset), movieScheduleBitset(_movieScheduleBitset), maxMoviesPerCat(_maxMoviesPerCat) {}

    __device__ bool operator()(int j)
    {
        if (*agendaBitset & *movieScheduleBitset)
        {
            atomicSub(&maxMoviesPerCat[moviesList[j].category - 1], 1);
            return true;
        }
        return false;
    }
};

struct max_cnt_comparator
{
    __device__ bool operator()(const thrust::tuple<int, int> &x, const thrust::tuple<int, int> &y)
    {
        return thrust::get<1>(x) > thrust::get<1>(y);
    }
};

void Thrust::generatePermutations()
{
    thrust::device_vector<int> d_maxMoviesPerCat(maxMoviesPerCat, maxMoviesPerCat + *categories);
    thrust::device_vector<int> d_movieScheduleBitset(movieScheduleBitset, movieScheduleBitset + *movies);
    thrust::device_vector<Movie> d_moviesList(moviesList, moviesList + *movies);

    thrust::device_vector<int> d_max_cnt(1, 0);
    thrust::device_vector<int> d_selectedMovie(1, 0);
    thrust::device_vector<int> d_cnt(1, 0);

    thrust::device_vector<int> d_results(1 << *movies, 0);

    for (int i = 0; i < (1 << *movies); i++)
    {
        thrust::fill(d_maxMoviesPerCat.begin(), d_maxMoviesPerCat.end(), *categories);
        thrust::fill(d_results.begin(), d_results.end(), 0);
        d_max_cnt[0] = 0;

        thrust::counting_iterator<int> first(0);
        thrust::counting_iterator<int> last = first + *movies;
        thrust::transform(first, last, d_results.begin(), movie_validity_checker(thrust::raw_pointer_cast(d_moviesList.data()), thrust::raw_pointer_cast(d_agendaBitset.data()), thrust::raw_pointer_cast(d_movieScheduleBitset.data()), thrust::raw_pointer_cast(d_maxMoviesPerCat.data())));

        int max_cnt = thrust::reduce(d_results.begin(), d_results.end());

        if (max_cnt > d_cnt[0])
        {
            d_selectedMovie[0] = i;
            d_cnt[0] = max_cnt;
        }
    }

    thrust::copy(d_selectedMovie.begin(), d_selectedMovie.end(), selectedMovie);
    thrust::copy(d_cnt.begin(), d_cnt.end(), cnt);
}
